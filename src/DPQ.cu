#include "hip/hip_runtime.h"

#include <DPQ.cuh>
#include <thrust/device_vector.h>
using namespace std;
/*this is the DPBF algorithm in Ding, Bolin, et al. "Finding top-k min-cost connected trees in databases." 2007 IEEE 23rd International Conference on Data Engineering. IEEE, 2007.

time complexity: O( 4^|Gamma| + 3^|Gamma||V|+ 2^|Gamma|* (|E| + |V|*(|Gamma| + log V)) )*/
typedef struct queue_element
{
	int v, p;
} queue_element;
typedef struct node
{
	int update = 0;
	int type;	  // =0: this is the single vertex v; =1: this tree is built by grown; =2: built by merge
	int cost, lb; // cost of this tree T(v,p);
	int u;		  // if this tree is built by grown, then it's built by growing edge (v,u);
	int p1, p2;	  // if this tree is built by merge, then it's built by merge T(v,p1) and T(v,p2);
} node;
int E, N, width, height;
int *lb1, *lb2;
int *visit, *queue_size, *tree_cost, *new_queue_size, *best, *dis, *in_queue_check, *dis_queue, *new_dis_queue;
int *all_pointer, *all_edge, *edge_cost, *non_overlapped_group_sets_IDs_gpu, *non_overlapped_group_sets_IDs_pointer_device;
dim3 blockPerGrid, threadPerGrid;
node *tree;
queue_element *queue_device, *new_queue_device;
std::vector<int> non_overlapped_group_sets_IDs_pointer_host;
int graph_v_of_v_idealID_DPBF_vertex_group_set_ID_gpu(int vertex, graph_v_of_v_idealID &group_graph,
													  std::unordered_set<int> &cumpulsory_group_vertices)
{

	/*time complexity: O(|Gamma|); this function returns the maximum group set ID for a single vertex*/
	// if group i have edge to v,v will give bit i value 1;
	int ID = 0;
	int pow_num = 0;
	for (auto it = cumpulsory_group_vertices.begin(); it != cumpulsory_group_vertices.end(); it++)
	{
		if (graph_v_of_v_idealID_contain_edge(group_graph, vertex, *it))
		{ // vertex is in group *it
			ID = ID + pow(2, pow_num);
		}
		pow_num++;
	}

	return ID;
}

void graph_v_of_v_idealID_DPBF_non_overlapped_group_sets_gpu(int group_sets_ID_range)
{

	/*this function calculate the non-empty and non_overlapped_group_sets_IDs of each non-empty group_set ID;

	time complexity: O(4^|Gamma|), since group_sets_ID_range=2^|Gamma|;

	the original DPBF code use the same method in this function, and thus has the same O(4^|Gamma|) complexity;*/

	std::vector<int> non_overlapped_group_sets_IDs; // <set_ID, non_overlapped_group_sets_IDs>
	int len = 0;
	for (int i = 1; i <= group_sets_ID_range; i++)
	{ // i is a nonempty group_set ID
		non_overlapped_group_sets_IDs_pointer_host[i] = len;
		for (int j = 1; j < group_sets_ID_range; j++)
		{ // j is another nonempty group_set ID
			if ((i & j) == 0)
			{ // i and j are non-overlapping group sets
				/* The & (bitwise AND) in C or C++ takes two numbers as operands and does AND on every bit of two numbers. The result of AND for each bit is 1 only if both bits are 1.
				https://www.programiz.com/cpp-programming/bitwise-operators */
				non_overlapped_group_sets_IDs.push_back(j);
				len++;
			}
		}
	}
	non_overlapped_group_sets_IDs_pointer_host[group_sets_ID_range + 1] = len;

	hipMallocManaged((void **)&non_overlapped_group_sets_IDs_gpu, sizeof(int) * len);
	hipMemcpy(non_overlapped_group_sets_IDs_gpu, non_overlapped_group_sets_IDs.data(), sizeof(int) * len, hipMemcpyHostToDevice);
	std::cout << "len= " << len << std::endl;
}

__global__ void Relax(queue_element *Queue_dev, int *queue_size, queue_element *new_queue_device, int *new_queue_size, int *sets_IDs, int *sets_IDS_pointer, int *edge, int *edge_cost, int *pointer, size_t pitch_node, size_t pitch_int, size_t pitch_dis, int *dis, node *tree, int *tree_cost, int inf, int *best, int full)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < *queue_size)
	{
		queue_element top_node = Queue_dev[idx];
		int v = top_node.v, p = top_node.p;
		int *row_v = (int *)((char *)tree_cost + v * pitch_int);
		node *row_node_v = (node *)((char *)tree + v * pitch_node);
		int x_slash = full - p;
		if (row_v[x_slash] != inf && row_v[p] != inf)
		{
			int new_best = row_v[x_slash] + row_v[p];
			atomicMin(best, new_best);
			if (new_best < *best)
			{
				atomicMin(&row_v[full], new_best);
				atomicMin(&row_node_v[full].cost, new_best);
				int check = atomicCAS(&row_node_v[full].update, 0, 1);
				if (!check)
				{
					int pos = atomicAdd(new_queue_size, 1);
					new_queue_device[pos] = {v, full};
				}
			}
		}
		
		if (row_v[p] > (*best) / 2)
		{
			return;
		}
		for (int i = pointer[v]; i < pointer[v + 1]; i++)
		{
			/*grow*/
			
			int u = edge[i];
			int cost_euv = edge_cost[i];
			int *row_u = (int *)((char *)tree_cost + u * pitch_int);
			int grow_tree_cost = row_v[p] + cost_euv;
			int old = atomicMin(&row_u[p], grow_tree_cost);
			node *row_node_u = (node *)((char *)tree + u * pitch_node);
			// get_lb(edge,edge_cost,pointer,pitch_node,pitch_int,pitch_dis,dis,tree,tree_cost,inf,best,full,grow_tree_cost,plb,v,p);
			
			atomicMin(&row_node_u[p].cost, grow_tree_cost);

			if (old >= grow_tree_cost && grow_tree_cost != inf)
			{
				row_node_u[p].type = 1;
				row_node_u[p].u = v;
				row_node_u[p].cost = grow_tree_cost;
				// enqueue operation
				int check = atomicCAS(&row_node_u[p].update, 0, 1);
				if (!check)
				{
					int pos = atomicAdd(new_queue_size, 1);
					new_queue_device[pos].v = u;
					new_queue_device[pos].p = p;
				}
			}
		}

		/*merge*/
		int p1 = p;
		for (auto it = sets_IDS_pointer[p1]; it < sets_IDS_pointer[p1 + 1]; it++)
		{
			int p2 = sets_IDs[it]; // p2 is not overlapped with p1
			int *row_v = (int *)((char *)tree_cost + v * pitch_int);
			int cost_Tvp1 = row_v[p1], cost_Tvp2 = row_v[p2];
			int p1_cup_p2 = p1 + p2;
			int merged_tree_cost = cost_Tvp1 + cost_Tvp2;
			// && merged_tree_cost < 2 / 3 * (*best)
			int old = atomicMin(&row_v[p1_cup_p2], merged_tree_cost);
			atomicMin(&row_node_v[p1_cup_p2].cost, merged_tree_cost);
			
			if (old >= merged_tree_cost && merged_tree_cost != inf)
			{ // O(3^|Gamma||V| comparisons in totel, see the DPBF paper)

				/*update T(v,p1_cup_p2) by merge T(v,p1) with T(v,v2)*/
				row_node_v[p1_cup_p2].type = 2;
				row_node_v[p1_cup_p2].p1 = p1;
				row_node_v[p1_cup_p2].p2 = p2;
				row_node_v[p1_cup_p2].cost = merged_tree_cost;

				if (merged_tree_cost < 0.667 * (*best))
				{
					int check = atomicCAS(&row_node_v[p1_cup_p2].update, 0, 1);
					if (!check)
					{
						int pos = atomicAdd(new_queue_size, 1);
						new_queue_device[pos].v = v;
						new_queue_device[pos].p = p1_cup_p2;
					}
				}
			}
		}
	}
}

__global__ void dis_init(int v, int *in_queue_check, int pitch_int, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		int *row_v = (int *)((char *)in_queue_check + v * pitch_int);
		row_v[idx] = 0;
	}
}
__global__ void dis_Relax(int *dis_queue, int *new_dis_queue, int *dis, int *in_queue_check, int *queue_size, int *new_queue_size, int *edge, int *edge_cost, int *pointer, size_t pitch_int, int source)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < *queue_size)
	{
		int u = dis_queue[idx];
		int *row_s = (int *)((char *)dis + source * pitch_int);
		int *row_in = (int *)((char *)in_queue_check + source * pitch_int);

		for (int i = pointer[u]; i < pointer[u + 1]; i++)
		{
			int v = edge[i];
			int new_w = row_s[u] + edge_cost[i];

			int old = atomicMin(&row_s[v], new_w);
			if (new_w < old)
			{
				int check = atomicCAS(&row_in[v], 0, 1);
				if (!check)
				{
					int pos = atomicAdd(new_queue_size, 1);
					new_dis_queue[pos] = v;
				}
			}
		}
	}
}
graph_hash_of_mixed_weighted DPBF_GPU(CSR_graph &graph, std::unordered_set<int> &cumpulsory_group_vertices, graph_v_of_v_idealID &group_graph, graph_v_of_v_idealID &input_graph,int *pointer1)
{
	double time_process = 0;
	auto begin = std::chrono::high_resolution_clock::now();
	E = graph.E_all, N = graph.V;
	int G = cumpulsory_group_vertices.size();
	all_edge = graph.all_edge, all_pointer = graph.all_pointer, edge_cost = graph.all_edge_weight;
	int group_sets_ID_range = pow(2, G) - 1;
	int inf = 1024;
	non_overlapped_group_sets_IDs_pointer_host.resize(group_sets_ID_range + 3);
	long long unsigned int problem_size = N * pow(2, cumpulsory_group_vertices.size());
	hipMallocManaged((void **)&non_overlapped_group_sets_IDs_pointer_device, sizeof(int) * (group_sets_ID_range + 3));
	hipMallocManaged((void **)&queue_size, sizeof(int));
	hipMallocManaged((void **)&new_queue_size, sizeof(int));
	hipMallocManaged((void **)&best, sizeof(int));
	hipMallocManaged((void **)&queue_device, problem_size * sizeof(queue_element));
	hipMallocManaged((void **)&new_queue_device, problem_size * sizeof(queue_element));
	hipMallocManaged((void **)&dis_queue, problem_size * sizeof(int));
	hipMallocManaged((void **)&in_queue_check, problem_size * sizeof(int));
	hipMallocManaged((void **)&new_dis_queue, problem_size * sizeof(int));
	graph_v_of_v_idealID_DPBF_non_overlapped_group_sets_gpu(group_sets_ID_range);
	hipMemcpy(non_overlapped_group_sets_IDs_pointer_device, non_overlapped_group_sets_IDs_pointer_host.data(), (group_sets_ID_range + 3) * sizeof(int), hipMemcpyHostToDevice);
	std::cout << "group range " << group_sets_ID_range << std::endl;
	threadPerGrid.x = THREAD_PER_BLOCK;
	blockPerGrid.x = (N + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
	width = group_sets_ID_range + 1, height = N;
	size_t pitch_node, pitch_int, pitch_vis, pitch_dis;
	node host_tree[height][width];
	int host_cost[height][width];
	int host_dis[G][N], host_f1[N][width], host_f2[N][width];
	queue_element host_queue[problem_size];
	hipMallocPitch(&dis, &pitch_dis, N * sizeof(int), G);
	hipMallocPitch(&in_queue_check, &pitch_vis, N * sizeof(int), G);
	hipMallocPitch(&tree, &pitch_node, width * sizeof(node), height);
	hipMallocPitch(&tree_cost, &pitch_int, width * sizeof(int), height);
	hipMallocPitch(&lb1, &pitch_int, width * sizeof(int), G);
	hipMallocPitch(&lb2, &pitch_int, width * sizeof(int), G);
	// hipMemset3D(devPitchedPtr, inf, extent);
	std::cout << "pitch " << pitch_node << " " << " width " << width << std::endl;
	auto end = std::chrono::high_resolution_clock::now();
	double runningtime = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / 1e9; // s
	time_process += runningtime;
	cout << "allocate cost time " << runningtime << endl;
	*best = inf;

	begin = std::chrono::high_resolution_clock::now();
	*queue_size = 0, *new_queue_size = 0;
	for (int v = 0; v < N; v++)
	{
		host_tree[v][0].cost = 0;
		host_cost[v][0] = 0;
		int group_set_ID_v = graph_v_of_v_idealID_DPBF_vertex_group_set_ID_gpu(v, group_graph, cumpulsory_group_vertices); /*time complexity: O(|Gamma|)*/
		for (int p = 1; p <= group_sets_ID_range; p++)
		{ // p is non-empty; time complexity: O(2^|Gamma|) //get all its subset ,which is required in next merge and grow steps
			host_tree[v][p].cost = inf;
			host_cost[v][p] = inf;
			if ((p | group_set_ID_v) == group_set_ID_v)
			{ // p represents a non-empty group set inside group_set_ID_v, including group_set_ID_v
				/*T(v,p)*/
				host_tree[v][p].cost = 0;
				host_tree[v][p].type = 0;
				host_cost[v][p] = 0;
				host_queue[*queue_size].v = v;
				host_queue[*queue_size].p = p;
				*queue_size += 1;
			}
		}
	}

	hipMemcpy2D(tree, pitch_node, host_tree, width * sizeof(node), width * sizeof(node), height, hipMemcpyHostToDevice);
	hipMemcpy2D(tree_cost, pitch_int, host_cost, width * sizeof(int), width * sizeof(int), height, hipMemcpyHostToDevice);
	hipMemcpy(queue_device, host_queue, *queue_size * sizeof(queue_element), hipMemcpyHostToDevice);
	// std::cout << "queue size init " << *queue_size << std::endl;
	// std::cout << "queue init " << std::endl;
	/* 	for (size_t i = 0; i < *queue_size; i++)
		{
			std::cout << " v " << queue_device[i].v << " p " << queue_device[i].p << "; ";
		} */
	cout << endl;
	int r = 0, process = 0;

	while (*queue_size != 0)
	{
		process += *queue_size;
		std::cout << "round " << r++ << std::endl;
		std::cout << "queue size " << *queue_size << std::endl;
		/* 		for (size_t i = 0; i < *queue_size; i++)
				{
					std::cout << " v " << queue_device[i].v << " p " << queue_device[i].p << "; ";
				} */
		cout << endl;
		Relax<<<(*queue_size + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(queue_device, queue_size, new_queue_device, new_queue_size, non_overlapped_group_sets_IDs_gpu,
																							 non_overlapped_group_sets_IDs_pointer_device, all_edge, edge_cost, all_pointer, pitch_node, pitch_int, pitch_dis, dis, tree, tree_cost, inf, best, group_sets_ID_range);
		hipDeviceSynchronize();
		/* 		hipMemcpy2D(host_tree, width * sizeof(node), tree, pitch_node, width * sizeof(node), height, hipMemcpyDeviceToHost);
				hipMemcpy2D(host_cost, width * sizeof(int), tree_cost, pitch_int, width * sizeof(int), height, hipMemcpyDeviceToHost);
				for (size_t i = 0; i < N; i++)
				{
					cout << i << " ";
					for (size_t j = 1; j <= group_sets_ID_range; j++)
					{
						cout << host_cost[i][j] << " ";
					}
					cout << endl;
				}
				cout<<"new size = "<<*new_queue_size<<endl; */
		*queue_size = *new_queue_size;
		*new_queue_size = 0;
		std::swap(queue_device, new_queue_device);
	}
	std::cout << "while over,process node  " << process << std::endl;
	*pointer1+=process;
	end = std::chrono::high_resolution_clock::now();
	runningtime = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / 1e9; // s
	cout << "gpu cost time " << runningtime << endl;
	hipMemcpy2D(host_tree, width * sizeof(node), tree, pitch_node, width * sizeof(node), height, hipMemcpyDeviceToHost);
	hipMemcpy2D(host_cost, width * sizeof(int), tree_cost, pitch_int, width * sizeof(int), height, hipMemcpyDeviceToHost);

	std::cout << "all copy complete ,now list cost " << std::endl;
	int min_cost = inf, min_node = -1;
	for (int i = 0; i < N; i++)
	{
		// cout << host_tree[i][group_sets_ID_range].cost << " ";
		if (host_cost[i][group_sets_ID_range] < min_cost)
		{
			min_cost = host_cost[i][group_sets_ID_range];
			min_node = i;
		}
	}

	std::cout << "root " << min_node << "cost " << min_cost << std::endl;
	graph_hash_of_mixed_weighted solution_tree;
	std::queue<std::pair<int, int>> waited_to_processed_trees; // <v, p>
	int root_v = min_node, root_p = group_sets_ID_range;
	waited_to_processed_trees.push({root_v, root_p});

	while (waited_to_processed_trees.size() > 0)
	{

		int v = waited_to_processed_trees.front().first, p = waited_to_processed_trees.front().second;
		waited_to_processed_trees.pop();

		graph_hash_of_mixed_weighted_add_vertex(solution_tree, v, 0);

		auto pointer_trees_v_p = host_tree[v][p];
		int form_type = pointer_trees_v_p.type;
		if (form_type == 0)
		{ // T(v,p) is a single vertex
		}
		else if (form_type == 1)
		{ // T(v,p) is formed by grow
			int u = host_tree[v][p].u;

			waited_to_processed_trees.push({u, p});
			/*insert (u,v); no need to insert weight of u here, which will be inserted later for T(u,p)*/
			int c_uv = graph_v_of_v_idealID_edge_weight(input_graph, u, v);
			graph_hash_of_mixed_weighted_add_edge(solution_tree, u, v, c_uv);
		}
		else
		{ // T(v,p) is formed by merge
			int p1 = host_tree[v][p].p1, p2 = host_tree[v][p].p2;

			waited_to_processed_trees.push({v, p1});
			waited_to_processed_trees.push({v, p2});
		}
	}
	return solution_tree;
}
